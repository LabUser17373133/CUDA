
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void VectAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}
int main()
{
    printf("Hello World From CPU!\n");
    float A_h[] = { 1, 2, 3 };
    float B_h[] = { 1, 2, 3 };
    float C_h[] = { 3, 2, 1 };
    
    
    float *A_d, *B_d, *C_d;
    hipMalloc((float**)&A_d, sizeof(A_h));
    hipMalloc((float**)&B_d, sizeof(B_h));
    hipMalloc((float**)&C_d, sizeof(C_h));
    hipMemcpy(A_d, A_h, sizeof(A_h), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, sizeof(B_h), hipMemcpyHostToDevice);
    hipMemcpy(C_d, C_h, sizeof(C_h), hipMemcpyHostToDevice);
    VectAdd <<<1, 3 >>> (A_d, B_d, C_d);
    hipMemcpy(A_h, A_d, sizeof(A_h), hipMemcpyDeviceToHost);
    hipMemcpy(B_h, B_d, sizeof(B_h), hipMemcpyDeviceToHost);
    hipMemcpy(C_h, C_d, sizeof(C_h), hipMemcpyDeviceToHost);
    printf("%f, %f, %f\n", C_h[0], C_h[1] , C_h[2]);
    hipDeviceReset();
    return 0;
}
